//https://github.com/Anghe-Mark-intelligence/ALAM-Affordance_Learning_automark/tree/main
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(err) { \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
        exit(-1); \
    } \
}

// GPU Kernel function: Forward pass
__global__ void forward_pass(float *input, float *weights, float *output, int num_input, int num_output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_output) {
        output[idx] = 0;
        // Compute output for the current neuron
        for (int i = 0; i < num_input; i++) {
            output[idx] += input[i] * weights[i * num_output + idx];
        }
    }
}

// GPU Kernel function: Backward pass
__global__ void backward_pass(float *input, float *weights, float *output, float *grad_input, float *grad_weights, int num_input, int num_output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_input) {
        grad_input[idx] = 0;
        // Compute gradients for input layer
        for (int i = 0; i < num_output; i++) {
            grad_input[idx] += output[i] * weights[idx * num_output + i];
        }
    }
    
    if (idx < num_output) {
        // Compute gradients for weights
        for (int i = 0; i < num_input; i++) {
            grad_weights[i * num_output + idx] = input[i] * output[idx];
        }
    }
}

// Main function
int main() {
    const int num_input = 4;  // Number of input neurons
    const int num_output = 3; // Number of output neurons
    float input[num_input] = {1.0f, 2.0f, 3.0f, 4.0f}; // Input data
    float weights[num_input * num_output] = {0.5f, -0.2f, 0.3f, 0.1f, -0.4f, 0.8f, -0.3f, 0.2f, 0.6f, 0.5f, -0.1f, 0.4f}; // Weights

    float *d_input, *d_weights, *d_output, *d_grad_input, *d_grad_weights;
    float output[num_output];

    hipError_t err;

    // Allocate memory on GPU
    err = hipMalloc((void**)&d_input, num_input * sizeof(float));
    CHECK_CUDA_ERROR(err);
    err = hipMalloc((void**)&d_weights, num_input * num_output * sizeof(float));
    CHECK_CUDA_ERROR(err);
    err = hipMalloc((void**)&d_output, num_output * sizeof(float));
    CHECK_CUDA_ERROR(err);
    err = hipMalloc((void**)&d_grad_input, num_input * sizeof(float));
    CHECK_CUDA_ERROR(err);
    err = hipMalloc((void**)&d_grad_weights, num_input * num_output * sizeof(float));
    CHECK_CUDA_ERROR(err);

    // Copy input data and weights from host to device (GPU)
    err = hipMemcpy(d_input, input, num_input * sizeof(float), hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(err);
    err = hipMemcpy(d_weights, weights, num_input * num_output * sizeof(float), hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(err);

    // Set the number of threads and blocks for execution
    int blockSize = 256;
    int numBlocks = (num_output + blockSize - 1) / blockSize;

    // Execute forward pass on GPU
    forward_pass<<<numBlocks, blockSize>>>(d_input, d_weights, d_output, num_input, num_output);
    err = hipGetLastError();
    CHECK_CUDA_ERROR(err);

    // Copy the output from device back to host
    err = hipMemcpy(output, d_output, num_output * sizeof(float), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err);

    // Print the result of the forward pass
    std::cout << "Output of forward pass:\n";
    for (int i = 0; i < num_output; i++) {
        std::cout << "output[" << i << "] = " << output[i] << "\n";
    }

    // Execute backward pass on GPU (for gradient computation)
    backward_pass<<<numBlocks, blockSize>>>(d_input, d_weights, d_output, d_grad_input, d_grad_weights, num_input, num_output);
    err = hipGetLastError();
    CHECK_CUDA_ERROR(err);

    // Clean up and free memory on GPU
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_output);
    hipFree(d_grad_input);
    hipFree(d_grad_weights);

    return 0;
}
